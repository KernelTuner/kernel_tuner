#include "hip/hip_runtime.h"
#define WIDTH 4096

__global__ void matmul_kernel(float *C, float *A, float *B) {

    __shared__ float sA[block_size_y][block_size_x];
    __shared__ float sB[block_size_y][block_size_x];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int x = blockIdx.x * block_size_x + tx;
    int y = blockIdx.y * block_size_y + ty;

    float sum = 0.0;
    int k,kb;

    for (k=0; k<WIDTH; k+=block_size_x) {
        __syncthreads();
        sA[ty][tx] = A[y*WIDTH+k+tx];
        sB[ty][tx] = B[(k+ty)*WIDTH+x];
        __syncthreads();

        for (kb=0; kb<block_size_x; kb++) {
            sum += sA[ty][kb] * sB[kb][tx];
        }

    }

    C[y*WIDTH+x] = sum;
}

