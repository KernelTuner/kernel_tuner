#include "hip/hip_runtime.h"
#define WIDTH 4096

__global__ void matmul_kernel(float *C, float *A, float *B) {
    int x = blockIdx.x * block_size_x + threadIdx.x;
    int y = blockIdx.y * block_size_y + threadIdx.y;
    float sum = 0.0;

    for (int k=0; k<WIDTH; k++) {
        sum += A[y*WIDTH+k] * B[k*WIDTH+x];
    }

    C[y*WIDTH+x] = sum;
}
