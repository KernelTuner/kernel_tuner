#include "hip/hip_runtime.h"
/**
 * The kernel is assumed to be tuned to each device by selecting
 * the best performing combination of thread block dimensions 
 * and tiling factors in X and Y. In this implementation tiling
 * in X increases the amount of work per thread block and tiling
 * in Y increases the amount of work per thread within the block. 
 * 
 * @author Ben van Werkhoven <b.vanwerkhoven@esciencecenter.nl>
 * 
 */

#define WIDTH 4096
/*
 * Optimized CUDA kernel for matrix multiplication
 *
 * This kernel is optimized according to the directions given
 * in: "Better performance at lower occupancy" by V. Volkov,
 * GPU Technology Conference, GTC 2010.
 *
 * The thread block dimensions (block_size_x, block_size_y) 
 * and tiling factors (tile_size_x, tile_size_y) are to be
 * tuned towards each GPU. This kernel assumes that
 * block_size_x = block_size_y * tile_size_y.
 *
 * The kernel computes C=A*B, where A, B, and C are square
 * matrices with height and width equal to WIDTH
 */
extern "C"
__global__ void matmul_kernel(float *C, float *A, float *B) {

    __shared__ float sA[block_size_y*tile_size_y][block_size_x];
    __shared__ float sB[block_size_y*tile_size_y][block_size_x * tile_size_x];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int x = blockIdx.x * block_size_x * tile_size_x + threadIdx.x;
    int y = blockIdx.y * block_size_y * tile_size_y + threadIdx.y;
    int k, kb;

    float sum[tile_size_y][tile_size_x];
    #pragma unroll
    for (int i = 0; i < tile_size_y; i++) {
        #pragma unroll
        for (int j = 0; j < tile_size_x; j++) {
            sum[i][j] = 0.0f;
        }
    }

    for (k = 0; k < WIDTH; k += block_size_x) {

        __syncthreads();
        #pragma unroll
        for (int i = 0; i < tile_size_y; i++) {
            sA[ty + block_size_y * i][tx] = A[(y+i*block_size_y) * WIDTH + k + tx];

            #pragma unroll
            for (int j = 0; j < tile_size_x; j++) {
                sB[ty + block_size_y * i][tx + j * block_size_x] = B[(k + ty + block_size_y * i) * WIDTH + x + j * block_size_x];
            }
        }
        __syncthreads();

        //compute
        #pragma unroll
        for (kb = 0; kb < block_size_x; kb++) {

            #pragma unroll
            for (int i = 0; i < tile_size_y; i++) {
            #pragma unroll
                for (int j = 0; j < tile_size_x; j++) {
                    sum[i][j] += sA[ty + block_size_y * i][kb] * sB[kb][tx + j * block_size_x];
                }
            }

        }

    }

    //store result
    #pragma unroll
    for (int i = 0; i < tile_size_y; i++) {
        #pragma unroll
        for (int j = 0; j < tile_size_x; j++) {
            C[y * WIDTH + x + block_size_y * i * WIDTH + j * block_size_x] = sum[i][j];
        }
    }

}



