
#include <hip/hip_runtime.h>
#define VERTICES 600

__constant__ float2 d_vertices[VERTICES];
__constant__ float d_slopes[VERTICES];

/*
 * This file contains the implementation of a CUDA Kernel for the
 * point-in-polygon problem using the crossing number algorithm
 *
 * The kernel cn_pnpoly is can be tuned using the following parameters:
 *    * block_size_x                any sensible thread block size
 *    * tile_size                   any sensible tile size value
 *    * between_method              any of [0, 1, 2, 3]
 *    * use_precomputed_slopes      enable or disable [0, 1]
 *    * use_method                  any of [0, 1]
 *
 * The kernel cn_pnpoly_naive is used for correctness checking.
 *
 * The algorithm used here is adapted from: 
 *     'Inclusion of a Point in a Polygon', Dan Sunday, 2001
 *     (http://geomalgorithms.com/a03-_inclusion.html)
 *
 * Author: Ben van Werkhoven <b.vanwerkhoven@esciencecenter.nl>
 */

#ifndef block_size_x
    #define block_size_x 256
#endif
#ifndef block_size_y
    #define block_size_y 1
#endif
#ifndef block_size_z
    #define block_size_z 1
#endif

#ifndef tile_size
    #define tile_size 1
#endif

#ifndef between_method
    #define between_method 0
#endif

#ifndef use_precomputed_slopes
    #define use_precomputed_slopes 0
#endif

#ifndef use_method
    #define use_method 0
#endif



/*
 * The is_between method returns a boolean that is True when the a is between c and b.
 * Since the kernel is instruction bound, the exact way in which you compute is_between
 * can have a dramatic effect on performance.
 * Note that the way the different methods handle coincidents of a with b and c differs slightly.
 */
__device__ __forceinline__ int is_between(float a, float b, float c) {
    #if between_method == 0
        return (b > a) != (c > a);
    #elif between_method == 1
        return ((b <= a) && (c > a)) || ((b > a) && (c <= a));
    #elif between_method == 2
        return ((a - b) == 0.0f) || ((a - b) * (a - c) < 0.0f);
    #elif between_method == 3
        //Interestingly enough method 3 exactly the same as method 2, only in a different order.
        //the performance difference between method 2 and 3 can be huge depending on all the other optimization parameters.
        return ((a - b) * (a - c) < 0.0f) || (a - b == 0.0f);
    #endif
}



/*
 * The Point-in-Polygon kernel
 */
__global__ void cn_pnpoly(int* bitmap, float2* points, int n) {
    int i = blockIdx.x * block_size_x * tile_size + threadIdx.x;
    if (i < n) {

        int c[tile_size];
        float2 lpoints[tile_size];
        #pragma unroll
        for (int ti=0; ti<tile_size; ti++) {
            c[ti] = 0;
            if (i+block_size_x*ti < n) {
                lpoints[ti] = points[i+block_size_x*ti];
            }
        }

        int k = VERTICES-1;

        for (int j=0; j<VERTICES; k = j++) {    // edge from vj to vk
            float2 vj = d_vertices[j]; 
            float2 vk = d_vertices[k]; 

            #if use_precomputed_slopes == 0
            float slope = (vk.x-vj.x) / (vk.y-vj.y);
            #elif use_precomputed_slopes == 1
            float slope = d_slopes[j];
            #endif

            #pragma unroll
            for (int ti=0; ti<tile_size; ti++) {

                float2 p = lpoints[ti];

                #if use_method == 0
                if (  is_between(p.y, vj.y, vk.y) &&         //if p is between vj and vk vertically
                     (p.x < slope * (p.y-vj.y) + vj.x)
                        ) {  //if p.x crosses the line vj-vk when moved in positive x-direction
                    c[ti] = !c[ti];
                }

                #elif use_method == 1
                //Same as method 0, but attempts to reduce divergence by avoiding the use of an if-statement.
                //Whether this is more efficient is data dependent because there will be no divergence using method 0, when none
                //of the threads within a warp evaluate is_between as true
                int b = is_between(p.y, vj.y, vk.y);
                c[ti] += b && (p.x < vj.x + slope * (p.y - vj.y));

                #endif


            }

        }

        #pragma unroll
        for (int ti=0; ti<tile_size; ti++) {
            //could do an if statement here if 1s are expected to be rare
            if (i+block_size_x*ti < n) {
                #if use_method == 0
                bitmap[i+block_size_x*ti] = c[ti];
                #elif use_method == 1
                bitmap[i+block_size_x*ti] = c[ti] & 1;
                #endif
            }
        }
    }

}


/*
 * The naive implementation is used for verifying correctness of the optimized implementation
 * And in turn we have a unit test written in Python for this kernel
 */
__global__ void cn_pnpoly_naive(int* bitmap, float2* points, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n) {
        int c = 0;
        float2 p = points[i];

        int k = VERTICES-1;

        for (int j=0; j<VERTICES; k = j++) {    // edge from v to vp
            float2 vj = d_vertices[j]; 
            float2 vk = d_vertices[k]; 

            float slope = (vk.x-vj.x) / (vk.y-vj.y);

            if ( (  (vj.y>p.y) != (vk.y>p.y)) &&            //if p is between vj and vk vertically
                    (p.x < slope * (p.y-vj.y) + vj.x) ) {   //if p.x crosses the line vj-vk when moved in positive x-direction
                c = !c;
            }
        }

        bitmap[i] = c; // 0 if even (out), and 1 if odd (in)
    }


}


