#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "pnpoly.cu"

#ifndef grid_size_x
    #define grid_size_x 1
#endif
#ifndef grid_size_y
    #define grid_size_y 1
#endif

/*
 * This function contains the host code for benchmarking the cn_pnpoly CUDA kernel
 * Including the time spent on data transfers between host and device memory
 *
 * This host code uses device mapped host memory to overlap communication
 * between host and device with kernel execution on the GPU. Because each input
 * is read only once and each output is written only once, this implementation
 * almost fully overlaps all communication and the kernel execution time dominates
 * the total execution time.
 *
 * The code has the option to precompute all polygon line slopes on the CPU and
 * reuse those results on the GPU, instead of recomputing them on the GPU all
 * the time. The time spent on precomputing these values on the CPU is also 
 * taken into account by the time measurement in the code below. 
 *
 * This code was written for use with the Kernel Tuner. See: 
 *      https://github.com/benvanwerkhoven/kernel_tuner
 *
 * Author: Ben van Werkhoven <b.vanwerkhoven@esciencecenter.nl>
 */
extern "C" float cn_pnpoly_host(int* bitmap, float2* points, float2* vertices, int n) {

    hipError_t err;

    #if use_precomputed_slopes == 1
    float *h_slopes;
    err = hipHostAlloc((void **)&h_slopes, VERTICES*sizeof(float), hipHostMallocMapped);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipHostAlloc: %s\n", hipGetErrorString(err));
    }
    #endif

    //create CUDA streams and events
    hipStream_t stream[1];
    err = hipStreamCreate(&stream[0]);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipStreamCreate: %s\n", hipGetErrorString(err));
    }
    hipEvent_t start;
    err = hipEventCreate(&start);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipEventCreate: %s\n", hipGetErrorString(err));
    }

    hipEvent_t stop;
    err = hipEventCreate(&stop);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipEventCreate: %s\n", hipGetErrorString(err));
    }

    //kernel parameters
    dim3 threads(block_size_x, block_size_y, block_size_z);
    dim3 grid(grid_size_x, grid_size_y);

    //start measuring time
    hipDeviceSynchronize();
    hipEventRecord(start, stream[0]);

    //transfer vertices to d_vertices
    err = hipMemcpyToSymbolAsync(HIP_SYMBOL(d_vertices), vertices, VERTICES*sizeof(float2), 0, hipMemcpyHostToDevice, stream[0]);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipMemcpyToSymbolAsync: %s\n", hipGetErrorString(err));
    }

    #if use_precomputed_slopes == 1
    //precompute the slopes and transfer to symbol d_slopes
    h_slopes[0] = (vertices[VERTICES-1].x - vertices[0].x) / (vertices[VERTICES-1].y - vertices[0].y);
    for (int i=1; i<VERTICES; i++) {
        h_slopes[i] = (vertices[i-1].x - vertices[i].x) / (vertices[i-1].y - vertices[i].y);
    }
    err = hipMemcpyToSymbolAsync(HIP_SYMBOL(d_slopes), h_slopes, VERTICES*sizeof(float), 0, hipMemcpyHostToDevice, stream[0]);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipMemcpyToSymbolAsync: %s\n", hipGetErrorString(err));
    }
    #endif

    //call the kernel
    cn_pnpoly<<<grid, threads, 0, stream[0]>>>(bitmap, points, n);  //using mapped memory

    //stop time measurement
    hipEventRecord(stop, stream[0]);
    hipDeviceSynchronize();
    float time = 0.0;
    hipEventElapsedTime(&time, start, stop);

    //cleanup
    #if use_precomputed_slopes == 1
    hipHostFree(h_slopes);
    #endif
    hipStreamDestroy(stream[0]);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) {
        const char *error_string = hipGetErrorString(err);
        if (strncmp("too many resources requested for launch", error_string, 10) == 0) {
            time = -1.0;
        } else {
            fprintf(stderr, "Error after CUDA kernel: %s\n", error_string);
            exit(1);
        }
    }

    return time; //ms
}
